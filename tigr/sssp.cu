#include "hip/hip_runtime.h"

#include "../shared/timer.hpp"
#include "../shared/tigr_utilities.hpp"
#include "../shared/graph.hpp"
#include "../shared/virtual_graph.hpp"
#include "../shared/globals.hpp"
#include "../shared/argument_parsing.hpp"
#include "../shared/gpu_error_check.cuh"




__global__ void kernel(unsigned int numParts, 
							unsigned int *nodePointer, 
							PartPointer *partNodePointer,
							unsigned int *edgeList, 
							unsigned int *dist, 
							bool *finished,
							bool *label1,
							bool *label2)
{
	int partId = blockDim.x * blockIdx.x + threadIdx.x;

	if(partId < numParts)
	{
		int id = partNodePointer[partId].node;
		int part = partNodePointer[partId].part;

		if(label1[id] == false)
			return;

		int sourceWeight = dist[id];

		int thisPointer = nodePointer[id];
		int degree = edgeList[thisPointer];

		int numParts;
		if(degree % Part_Size == 0)
			numParts = degree / Part_Size ;
		else
			numParts = degree / Part_Size + 1;
		
		int end;
		int w8;
		int finalDist;
		int ofs = thisPointer + 2*part +1;

		for(int i=0; i<Part_Size; i++)
		{
			if(part + i*numParts >= degree)
				break;
			end = ofs + i*numParts*2;
			w8 = end + 1;
			finalDist = sourceWeight + edgeList[w8];
			if(finalDist < dist[edgeList[end]])
			{
				atomicMin(&dist[edgeList[end]] , finalDist);
				*finished = false;

				label2[edgeList[end]] = true;
			}
		}
	
	}
}

__global__ void clearLabel(bool *label, unsigned int size)
{
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < size)
		label[id] = false;
}

int main(int argc, char** argv)
{	
	ArgumentParser arguments(argc, argv, true, false);
	
	Graph graph(arguments.input, true);
	graph.ReadGraph();

	VirtualGraph vGraph(graph);
	
	vGraph.MakeGraph();
	
	uint num_nodes = graph.num_nodes;
	uint num_edges = graph.num_edges;

	if(arguments.hasDeviceID)
		hipSetDevice(arguments.deviceID);
	
	hipFree(0);
	
	unsigned int *dist;
	dist  = new unsigned int[num_nodes];

	bool *label1;
	bool *label2;
	label1 = new bool[num_nodes];
	label2 = new bool[num_nodes];
	
	for(int i=0; i<num_nodes; i++)
	{
			dist[i] = DIST_INFINITY;
			label1[i] = false;
			label2[i] = false;
	}
	
	dist[arguments.sourceNode] = 0;
	label1[arguments.sourceNode] = true;
	

	uint *d_nodePointer;
	uint *d_edgeList;
	uint *d_dist;
	PartPointer *d_partNodePointer; 
	bool *d_label1;
	bool *d_label2;
	
	bool finished;
	bool *d_finished;

	gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_dist, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label1, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label2, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));

	gpuErrorcheck(hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (2*num_edges + num_nodes) * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_dist, dist, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label1, label1, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_label2, label2, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer), hipMemcpyHostToDevice));
	
	Timer t;
	t.Start();

	int itr = 0;
	do
	{
		itr++;
		finished = true;
		gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
		if(itr % 2 == 1)
		{
			kernel<<< vGraph.numParts/512 + 1 , 512 >>>(vGraph.numParts, 
														d_nodePointer,
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														d_label1,
														d_label2);
			clearLabel<<< num_nodes/512 + 1 , 512 >>>(d_label1, num_nodes);
		}
		else
		{
			kernel<<< vGraph.numParts/512 + 1 , 512 >>>(vGraph.numParts, 
														d_nodePointer, 
														d_partNodePointer,
														d_edgeList, 
														d_dist, 
														d_finished,
														d_label2,
														d_label1);
			clearLabel<<< num_nodes/512 + 1 , 512 >>>(d_label2, num_nodes);
		}

		gpuErrorcheck( hipPeekAtLastError() );
		gpuErrorcheck( hipDeviceSynchronize() );	
		
		gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
		

	} while (!(finished));
	
	cout << "Number of iterations = " << itr << endl;

	
	float runtime = t.Finish();
	cout << "Processing finished in " << runtime << " (ms).\n";
	
	gpuErrorcheck(hipMemcpy(dist, d_dist, num_nodes*sizeof(unsigned int), hipMemcpyDeviceToHost));

	utilities::PrintResults(dist, 30);
			
	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist, num_nodes);

	gpuErrorcheck(hipFree(d_nodePointer));
	gpuErrorcheck(hipFree(d_edgeList));
	gpuErrorcheck(hipFree(d_dist));
	gpuErrorcheck(hipFree(d_finished));
	gpuErrorcheck(hipFree(d_label1));
	gpuErrorcheck(hipFree(d_label2));
	gpuErrorcheck(hipFree(d_partNodePointer));

}
